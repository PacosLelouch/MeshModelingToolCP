#include "ExportCommon.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace CUDAExport
{    
    DLL_EXPORT void hostMalloc(void** hstPtr, size_t size)
    {
        hipHostMalloc(hstPtr, size);
    }

    DLL_EXPORT void hostFree(void* hstPtr)
    {
        hipHostFree(hstPtr);
    }

    DLL_EXPORT void deviceMalloc(void** devPtr, size_t size)
    {
        hipMalloc(devPtr, size);
    }

    DLL_EXPORT void deviceFree(void* devPtr)
    {
        hipFree(devPtr);
    }

    DLL_EXPORT void deviceSync()
    {
        hipDeviceSynchronize();
    }

    DLL_EXPORT void deviceToHostMemcpy(void* hstDst, void* devSrc, size_t size)
    {
        hipMemcpy(hstDst, devSrc, size, hipMemcpyDeviceToHost);
    }

    DLL_EXPORT void hostToHostMemcpy(void* hstDst, void* hstSrc, size_t size)
    {
        hipMemcpy(hstDst, hstSrc, size, hipMemcpyHostToHost);
    }

    DLL_EXPORT void deviceToDeviceMemcpy(void* devDst, void* devSrc, size_t size)
    {
        hipMemcpy(devDst, devSrc, size, hipMemcpyDeviceToDevice);
    }

    DLL_EXPORT void hostToDeviceMemcpy(void* devDst, void* hstSrc, size_t size)
    {
        hipMemcpy(devDst, hstSrc, size, hipMemcpyHostToDevice);
    }
}
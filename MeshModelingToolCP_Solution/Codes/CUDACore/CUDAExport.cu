#include "CUDAExportCommon.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace CUDAExport
{    
    CUDA_LIB_EXPORT void hostMalloc(void** hstPtr, size_t size)
    {
        hipHostMalloc(hstPtr, size);
    }

    CUDA_LIB_EXPORT void hostFree(void* hstPtr)
    {
        hipHostFree(hstPtr);
    }

    CUDA_LIB_EXPORT void deviceMalloc(void** devPtr, size_t size)
    {
        hipMalloc(devPtr, size);
    }

    CUDA_LIB_EXPORT void deviceFree(void* devPtr)
    {
        hipFree(devPtr);
    }

    CUDA_LIB_EXPORT void deviceSync()
    {
        hipDeviceSynchronize();
    }

    CUDA_LIB_EXPORT void deviceToHostMemcpy(void* hstDst, void* devSrc, size_t size)
    {
        hipMemcpy(hstDst, devSrc, size, hipMemcpyDeviceToHost);
    }

    CUDA_LIB_EXPORT void hostToHostMemcpy(void* hstDst, void* hstSrc, size_t size)
    {
        hipMemcpy(hstDst, hstSrc, size, hipMemcpyHostToHost);
    }

    CUDA_LIB_EXPORT void deviceToDeviceMemcpy(void* devDst, void* devSrc, size_t size)
    {
        hipMemcpy(devDst, devSrc, size, hipMemcpyDeviceToDevice);
    }

    CUDA_LIB_EXPORT void hostToDeviceMemcpy(void* devDst, void* hstSrc, size_t size)
    {
        hipMemcpy(devDst, hstSrc, size, hipMemcpyHostToDevice);
    }
}
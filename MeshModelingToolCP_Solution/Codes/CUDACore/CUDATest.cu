#include "hip/hip_runtime.h"
#include "CUDATest.h"
#include "CUDAUtil.cuh"
#include <stdio.h>
#include <string.h>
#include <string>

#if defined _WIN32 || defined _WIN64
#include <Windows.h>

#else
#include <stdio.h>
#endif

using namespace CUDAUtil;

namespace CUDATest 
{
__global__ static void get_cude_arch_k(int* d_arch)
{

#if defined(__CUDA_ARCH__)
    *d_arch = __CUDA_ARCH__;
#else
    *d_arch = 0;
#endif
}

inline int cuda_arch()
{
    int* d_arch = 0;
    CUDA_ERROR(hipMalloc((void**)&d_arch, sizeof(int)));
    get_cude_arch_k<<<1, 1>>>(d_arch);
    int h_arch = 0;
    CUDA_ERROR(
        hipMemcpy(&h_arch, d_arch, sizeof(int), hipMemcpyDeviceToHost));
    hipFree(d_arch);
    return h_arch;
}

template<typename TCHAR>
static TCHAR* getFrac(TCHAR* dst, double f, int precision) 
{
    //int result = 0;
    double f1 = f;
    int i1 = static_cast<int>(f);
    size_t cursor = 0;
    for (int _ = 0; _ < precision; ++_) {
        f1 = (f1 - (double)i1) * 10.;
        dst[cursor++] = '0' + (int)f1;
        i1 = static_cast<int>(f1);
    }
    dst[cursor] = '\0';
    return dst;
}

#ifdef UNICODE
using CHAR_T = WCHAR;
using LPSTR_T = LPWSTR;
#define mbstowcs_t(A,B,C) mbstowcs(A,B,C)
#else
using CHAR_T = CHAR;
using LPSTR_T = LPSTR;
#define mbstowcs_t(A,B,C) strcpy(A,B)
#endif

hipDeviceProp_t cuda_query(const int dev, bool quiet = false, std::string* outStr = nullptr)
{
    hipDeviceProp_t devProp;
    CHAR_T message[16384]{ 0 };
    //memset(message, 0, sizeof(message));
    LPSTR_T cursor = message;
    //cursor += wsprintf(cursor, TEXT("----------------CUDATestFunction------------------\n"));
    // Various query about the device we are using
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0) 
    {
        cursor += wsprintf(cursor, 
            TEXT("cuda_query() device count = 0 i.e., there is not") 
            TEXT(" a CUDA-supported GPU!!!\n"));
    }
    else
    {
        hipSetDevice(dev);

        CUDA_ERROR(hipGetDeviceProperties(&devProp, dev));

        cursor += wsprintf(cursor, TEXT("Total number of device: %d\n"), deviceCount);
        cursor += wsprintf(cursor, TEXT("Using device Number: %d\n"), dev);
        CHAR_T devName[256]{ 0 };
        mbstowcs_t(devName, devProp.name, strlen(devProp.name));
        cursor += wsprintf(cursor, TEXT("Device name: %s\n"), devName);
        cursor += wsprintf(cursor, TEXT("Compute Capability: %d.%d\n"), (int)devProp.major,
            (int)devProp.minor);
        float gmem = (float)devProp.totalGlobalMem / 1048576.0f;
        CHAR_T frac0[16]{ 0 };
        CHAR_T frac1[16]{ 0 };
        cursor += wsprintf(cursor, TEXT("Total amount of global memory (MB): %d.%s\n"), 
            (int)gmem, getFrac(frac0, gmem, 1));
        cursor += wsprintf(cursor, TEXT("%d Multiprocessors, %d CUDA Cores/MP: %d CUDA Cores\n"), 
            devProp.multiProcessorCount,
            convert_SMV_to_cores(devProp.major, devProp.minor),
            convert_SMV_to_cores(devProp.major, devProp.minor) *
            devProp.multiProcessorCount);
        float GPUMaxClockRateMHz = devProp.clockRate * 1e-3f;
        float GPUMaxClockRateGHz = devProp.clockRate * 1e-6f;
        cursor += wsprintf(cursor, TEXT("GPU Max Clock rate: %d.%s MHz (%d.%s GHz)\n"), 
            (int)GPUMaxClockRateMHz, getFrac(frac0, GPUMaxClockRateMHz, 2),
            (int)GPUMaxClockRateGHz, getFrac(frac1, GPUMaxClockRateGHz, 2));
        float memMaxClockRateMHz = devProp.memoryClockRate * 1e-3f;
        cursor += wsprintf(cursor, TEXT("Memory Clock rate: %d.%s Mhz\n"), 
            (int)memMaxClockRateMHz, getFrac(frac0, memMaxClockRateMHz, 2));
        cursor += wsprintf(cursor, TEXT("Memory Bus Width:  %d-bit\n"), devProp.memoryBusWidth);
        const double maxBW = 2.0 * devProp.memoryClockRate *
            (devProp.memoryBusWidth / 8.0) / 1.0E6;

        cursor += wsprintf(cursor, TEXT("Peak Memory Bandwidth: %d.%s(GB/s)\n"), (int)maxBW, getFrac(frac0, maxBW, 2));
        cursor += wsprintf(cursor, TEXT("Kernels compiled for compute capability: %d"), 
            cuda_arch());
    }

    if (!quiet) {
#if defined _WIN32 || defined _WIN64
        MessageBox(NULL, message, TEXT("Third Party Plugin CUDA Test"), MB_OK);
#else
        wprintf(TEXT("%s"), message);
#endif
    }

    if (outStr)
    {
        CHAR messageANSI[16384]{ 0 };
        if (CharToOem(message, messageANSI))
        {
            outStr->assign(messageANSI);
        }
    }

    return devProp;
}

void CUDATest::CUDATestFunction(bool quiet, std::string* outStr) 
{
    cuda_query(0, quiet, outStr);
}
}
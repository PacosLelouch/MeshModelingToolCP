#include "ExportCommon.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace CUDAExport
{    
    MAYA_EXPORT void hostMalloc(void** hstPtr, size_t size)
    {
        hipHostMalloc(hstPtr, size);
    }

    MAYA_EXPORT void hostFree(void* hstPtr)
    {
        hipHostFree(hstPtr);
    }

    MAYA_EXPORT void deviceMalloc(void** devPtr, size_t size)
    {
        hipMalloc(devPtr, size);
    }

    MAYA_EXPORT void deviceFree(void* devPtr)
    {
        hipFree(devPtr);
    }

    MAYA_EXPORT void deviceSync()
    {
        hipDeviceSynchronize();
    }

    MAYA_EXPORT void deviceToHostMemcpy(void* hstDst, void* devSrc, size_t size)
    {
        hipMemcpy(hstDst, devSrc, size, hipMemcpyDeviceToHost);
    }

    MAYA_EXPORT void hostToHostMemcpy(void* hstDst, void* hstSrc, size_t size)
    {
        hipMemcpy(hstDst, hstSrc, size, hipMemcpyHostToHost);
    }

    MAYA_EXPORT void deviceToDeviceMemcpy(void* devDst, void* devSrc, size_t size)
    {
        hipMemcpy(devDst, devSrc, size, hipMemcpyDeviceToDevice);
    }

    MAYA_EXPORT void hostToDeviceMemcpy(void* devDst, void* hstSrc, size_t size)
    {
        hipMemcpy(devDst, hstSrc, size, hipMemcpyHostToDevice);
    }
}